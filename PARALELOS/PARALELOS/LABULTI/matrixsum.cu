#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__
void matrixSum(float * A, float * B, float * C, int n){
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if(col < n && row < n){
		C[row * n + col] = A[row * n + col] + B[row * n + col];
	}
}


void hostMatrixSum(float * A, float * B, float * C, int n){
	int size = sizeof(float) * n * n;
	float * d_A;
	float * d_B;
	float * d_C;
	hipMalloc((void **) &d_A, size);
	hipMalloc((void **) &d_B, size);
	hipMalloc((void **) &d_C, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	dim3 dimGrid(ceil(n/16.0),ceil(n/16.0),1);
	dim3 dimBlock(16,16,1);
	matrixSum<<<dimGrid,dimBlock>>>(d_A,d_B,d_C,n);
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	hipFree(d_A);
}

int main(int argv, char ** argc){
	if(argv != 2){
		printf("Faltan Argumentos <tamMatrix>\n");
		return 0;
	}
	int n = atoi(argc[1]);
	float * A = (float *) malloc(sizeof(float) * n * n);
	for(int i = 0; i < n * n; i++) A[i] = 1;
	float * B = (float *) malloc(sizeof(float) * n * n);
	for(int i = 0; i < n * n; i++) B[i] = 1;
	float * C = (float *) malloc(sizeof(float) * n * n);
	hostMatrixSum(A,B,C,n);
	//for(int i = 0; i < n * n; i++) printf("%f ", C[i]);
}
