#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define NUM_THREADS 16
#define BLUR_SIZE 3


__global__ void blur(int * img, int * res, int altura, int ancho){
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if(col < ancho && row < altura){
		int pixVal = 0;
		int pixeles = 0;
		for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow){
			for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol){
				int curRow = row + blurRow;
				int curCol = col + blurCol;
			
				if(curRow > -1 && curRow < altura && curCol > -1 && curCol < ancho){
					pixVal += img[curRow * ancho + curCol];
					pixeles++;
				}
			}
		}
		res[row * ancho + col] = (unsigned char)(pixVal / pixeles);
	}
	
}
void hostblur(int ** img, int ** res, int altura, int ancho){
	int size = sizeof(int) * altura * ancho;
	int * d_img;
	int * d_res;
	for(int i = 0; i < 3; i++){
		hipMalloc((void**) &d_img, size);
		hipMalloc((void**) &d_res, size);
		hipMemcpy(d_img, img[i], size, hipMemcpyHostToDevice);
		dim3 dimGrid(ceil(ancho/NUM_THREADS),ceil(altura/NUM_THREADS),1);
		dim3 dimBlock(NUM_THREADS,NUM_THREADS,1);
		blur<<<dimGrid,dimBlock>>>(d_img,d_res,altura,ancho);
		hipMemcpy(res[i], d_res, size, hipMemcpyDeviceToHost);
		hipFree(d_img);
		hipFree(d_res);
	}
}


int ** loadImage(char * file, int * altura, int * ancho){
	FILE * archivo = fopen(file,"r");
	char * line;
	size_t len = 0;
	getline(&line,&len,archivo);
	*altura = atoi(line);
	getline(&line,&len,archivo);
	*ancho = atoi(line);
	int ** res = (int **) malloc(sizeof(void *) * 3);
	for(int i = 0; i < 3; i++){
		res[i] = (int *) malloc(sizeof(int) * (*ancho) * (*altura));
		for(int j = 0; j < *altura; j++){
			for(int k = 0; k < *ancho; k++){
				getline(&line,&len,archivo);
				res[i][j * (*ancho) + k] = atoi(line);
			}
		}
	}
	fclose(archivo);
	return res;
}

void saveImage(int ** res, char * file, int altura, int ancho){
	FILE * archivo = fopen(file,"w");
	fprintf(archivo, "%d\n", altura);
	fprintf(archivo, "%d\n", ancho);
	for(int k = 0; k < 3; k++){
		for(int i = 0; i < altura; i++){
			for(int j = 0; j < ancho; j++){
				fprintf(archivo, "%d\n", res[k][i * ancho + j]);
			}	
		}
	}
	fclose(archivo);
}



int main(int argv, char ** argc){
	if(argv != 2){
		printf("Faltan Argumentos <img>\n");
		return 0;
	}
	char * fileImg = argc[1];
    char * programImg = "./img ";
	char * temp = " 0 >> temp"; 
	char * comp = (char *) malloc(strlen(programImg) + strlen(fileImg) + strlen(temp));
	comp = strcpy(comp, programImg);
	comp = strcat(comp, fileImg);
	comp = strcat(comp, temp);
	printf("%s\n", comp);
	system(comp);
	int altura = 0;
	int ancho = 0;
	int ** img = loadImage("temp",&altura, &ancho);
	int ** res = (int **) malloc(sizeof(void *) * 3);
	for(int i = 0; i < 3; i++){
		res[i] = (int *) malloc(sizeof(int) * ancho) * altura));
	}
	system("rm temp");
	hostblur(img,res,altura,ancho);
	saveImage(res,"res.txt",altura,ancho);
	system("./img res.txt 2");
	system("convert res.txt res.jpg");
	system("rm res.txt");
}
